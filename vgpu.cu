
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <chrono>
#include <cstdlib>
#include <map>
#include <vector>
#include <random>
#include <algorithm>
#include <cfloat>
#include <math.h>
#include <sys/time.h>


#define THREADS_PER_BLOCK 256

__global__ void voronoi_d (int *imageArray, int *points, int imageSize, int numPoints) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    double minDistance = DBL_MAX;
    int minPoint = -1;
    for (int k=0; k<numPoints; k++) {
        double distance = sqrt(pow((double) (x % imageSize - points[k + numPoints]), 2.0) + pow((double) (x / imageSize - points[k]), 2.0));
        if (distance < minDistance) {
            minDistance = distance;
            minPoint = k;
        }
    }
    imageArray[x] = minPoint;
}

extern void gpuVoronoi(int *imageArray_h, int *points_h, int imageSize, int numPoints)
{
    printf("starting host code \n");
    struct timeval start, end;

    int *imageArray;
    int *points;

	hipMalloc ((void**) &imageArray, sizeof(int) * imageSize * imageSize);
	hipMalloc ((void**) &points, sizeof(int) * numPoints * 2);
    hipMemcpy (points, points_h, sizeof(int) * numPoints * 2, hipMemcpyHostToDevice);

    gettimeofday(&start, NULL);

    voronoi_d <<< ceil((float) imageSize*imageSize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (imageArray, points, imageSize, numPoints);
    hipDeviceSynchronize();

    gettimeofday(&end, NULL);

    long seconds = (end.tv_sec - start.tv_sec);
    long micros = ((seconds * 1000000) + end.tv_usec) - (start.tv_usec);

    printf("Processing time elpased is %zu seconds or %zu micros\n", seconds, micros);

    hipError_t err = hipGetLastError();
    printf("CUDA error: %s\n", hipGetErrorString(err));

    hipMemcpy (imageArray_h, imageArray, sizeof(int) * imageSize * imageSize, hipMemcpyDeviceToHost);
    hipFree (imageArray);
    hipFree (points);

    printf("ending host code \n");
}

