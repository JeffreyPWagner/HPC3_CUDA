
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <chrono>
#include <cstdlib>
#include <map>
#include <vector>
#include <random>
#include <algorithm>
#include <cfloat>
#include <math.h>


#define THREADS_PER_BLOCK 256

__global__ void voronoi_d (int *imageArray, int *points, int imageSize, int numPoints) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    double minDistance = 1000000;
    int minPoint = -1;
    for (int k=0; k<numPoints; k++) {
        double distance = sqrt(pow((double) (x % imageSize - points[k + numPoints]), 2.0) + pow((double) (x / imageSize - points[k]), 2.0));
        if (distance < minDistance) {
            minDistance = distance;
            minPoint = k;
        }
    }
    imageArray[x] = minPoint;
}

extern void gpuVoronoi(int *imageArray_h, int *points_h, int imageSize, int numPoints)
{
    printf("starting host code \n");

    int *imageArray;
    int *points;

	hipMalloc ((void**) &imageArray, sizeof(int) * imageSize * imageSize);
	hipMalloc ((void**) &points, sizeof(int) * numPoints * 2);
    hipMemcpy (points, points_h, sizeof(int) * numPoints * 2, hipMemcpyHostToDevice);

    voronoi_d <<< ceil((float) imageSize*imageSize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (imageArray, points, imageSize, numPoints);

    hipError_t err = hipGetLastError();
    printf("CUDA error: %s\n", hipGetErrorString(err));

    hipMemcpy (imageArray_h, imageArray, sizeof(int) * imageSize * imageSize, hipMemcpyDeviceToHost);
    hipFree (imageArray);
    hipFree (points);

    printf("ending host code \n");
}

